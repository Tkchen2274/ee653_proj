
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_cuda() {
	printf("Hello from CUDA! Thread %d in block %d\n", threadIdx.x, blockIdx.x);
}
int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hello_cuda<<<1, 10>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %.3f ms\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
